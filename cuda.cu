// Include required header files

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <string.h>
#include <time.h>
#include <stdlib.h>
#include <math.h>

#include <hip/hip_runtime_api.h>

// Constants used for determining what print statements are executed.
#define PRINT_ERR 0
#define PRINT_TIME 0
#define PRINT_RESULT 1

// Function prototypes
char *create_input(int);
char *sobel(char *);

__global__ void VectorComputeSobel(char *, char *);

// Code for calculating elapsed time between two points in the code adpated from PCA
//    (EEL 6763) Lecture - Spring 2019 semester.
typedef double ttype;

ttype tdiff(struct timespec a, struct timespec b)
{
  ttype dt = (( b.tv_sec - a.tv_sec) + ( b.tv_nsec - a.tv_nsec ) / 1E9);
  return dt;
}

struct timespec now()
{
  struct timespec t;
  clock_gettime(CLOCK_REALTIME, &t);
  return t;
}

// Create a vector of size image_size filled wit random integers between 0 and 255, inclusive.
char *create_input(int image_size) {
  
  int i;

  char *image;

  image = (char *) malloc(image_size*sizeof(char));

  for (i = 0; i < image_size; ++i) {
    image[i] = (int) (rand() % 256);
  }

  return image;

}

// Create a vector of size image_size filled with zeros.
char *initialize_output(int image_size) {

  int i;

  char *image;

  image = (char *) malloc(image_size*sizeof(char));

  for (i = 0; i < image_size; ++i) {
    image[i] = 0;
  }

  return image;

}

// Serial implementation of the 3x3 Sobel filter.
char *sobel_serial(int width, char *image) {

  int i;
  int image_size = width*width;

  char *sobel_image;

  int Gx[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
  int Gy[9] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};

  int S_1, S_2;
  double G_b;

  sobel_image = (char *) malloc(image_size*sizeof(char));

  for (i = 0; i < image_size; ++i) {

    if (i/width == 0 || i/width == width-1 || i%width == 0 || i%width == width-1) {
      sobel_image[(i/width)*width + (i%width)] = 0;
      continue;
    }

    S_1 = (image[((i/width)-1)*width + ((i%width)-1)] & 0x000000FF)*Gx[0] +
          //(image[i-1][j  ] & 0x000000FF)*Gx[1] +
          (image[((i/width)-1)*width + ((i%width)+1)] & 0x000000FF)*Gx[2] +
          (image[((i/width)  )*width + ((i%width)-1)] & 0x000000FF)*Gx[3] +
          //(image[i  ][j  ] & 0x000000FF)*Gx[4] +
          (image[((i/width)  )*width + ((i%width)+1)] & 0x000000FF)*Gx[5] +
          (image[((i/width)+1)*width + ((i%width)-1)] & 0x000000FF)*Gx[6] +
          //(image[i+1][j  ] & 0x000000FF)*Gx[7] +
          (image[((i/width)+1)*width + ((i%width)+1)] & 0x000000FF)*Gx[8];

    S_2 = (image[((i/width)-1)*width + ((i%width)-1)] & 0x000000FF)*Gy[0] +
          (image[((i/width)-1)*width + ((i%width)  )] & 0x000000FF)*Gy[1] +
          (image[((i/width)-1)*width + ((i%width)+1)] & 0x000000FF)*Gy[2] +
          //(image[i  ][j-1] & 0x000000FF)*Gy[3] +
          //(image[i  ][j  ] & 0x000000FF)*Gy[4] +
          //(image[i  ][j+1] & 0x000000FF)*Gy[5] +
          (image[((i/width)+1)*width + ((i%width)-1)] & 0x000000FF)*Gy[6] +
          (image[((i/width)+1)*width + ((i%width)  )] & 0x000000FF)*Gy[7] +
          (image[((i/width)+1)*width + ((i%width)+1)] & 0x000000FF)*Gy[8];
      
    G_b = sqrt((double) (S_1*S_1 + S_2*S_2));
      
    if (G_b < 0.0)
      G_b = 0.0;

    sobel_image[(i/width)*width + (i%width)] = (int) round(G_b);
  }

  return sobel_image;

}

// GPU implementation of the 3x3 Sobel filter.
__global__ void VectorComputeSobel(char *image, char *sobel_image, int width) {

  int Gx[9] = {-1, 0, 1, -2, 0, 2, -1, 0, 1};
  int Gy[9] = {-1, -2, -1, 0, 0, 0, 1, 2, 1};

  int i = threadIdx.x + (blockIdx.x * blockDim.x);

  int S_1, S_2;
  double G_b;

  if (i/width == 0 || i/width == width-1 || i%width == 0 || i%width == width-1) {
      sobel_image[(i/width)*width + (i%width)] = 0;
      return;
  }

  S_1 = (image[((i/width)-1)*width + ((i%width)-1)] & 0x000000FF)*Gx[0] +
        //(image[i-1][j  ] & 0x000000FF)*Gx[1] +
        (image[((i/width)-1)*width + ((i%width)+1)] & 0x000000FF)*Gx[2] +
        (image[((i/width)  )*width + ((i%width)-1)] & 0x000000FF)*Gx[3] +
        //(image[i  ][j  ] & 0x000000FF)*Gx[4] +
        (image[((i/width)  )*width + ((i%width)+1)] & 0x000000FF)*Gx[5] +
        (image[((i/width)+1)*width + ((i%width)-1)] & 0x000000FF)*Gx[6] +
        //(image[i+1][j  ] & 0x000000FF)*Gx[7] +
        (image[((i/width)+1)*width + ((i%width)+1)] & 0x000000FF)*Gx[8];

  S_2 = (image[((i/width)-1)*width + ((i%width)-1)] & 0x000000FF)*Gy[0] +
        (image[((i/width)-1)*width + ((i%width)  )] & 0x000000FF)*Gy[1] +
        (image[((i/width)-1)*width + ((i%width)+1)] & 0x000000FF)*Gy[2] +
        //(image[i  ][j-1] & 0x000000FF)*Gy[3] +
        //(image[i  ][j  ] & 0x000000FF)*Gy[4] +
        //(image[i  ][j+1] & 0x000000FF)*Gy[5] +
        (image[((i/width)+1)*width + ((i%width)-1)] & 0x000000FF)*Gy[6] +
        (image[((i/width)+1)*width + ((i%width)  )] & 0x000000FF)*Gy[7] +
        (image[((i/width)+1)*width + ((i%width)+1)] & 0x000000FF)*Gy[8];

  G_b = sqrt((double) (S_1*S_1 + S_2*S_2));

  if (G_b < 0.0)
    G_b = 0.0;

  sobel_image[(i/width)*width + (i%width)] = (int) round(G_b);

  return;

}

int main(int argc, char *argv[])
{
  // Receive command line arguments.
  if (argc != 2) {
    printf("usage: ./hw4_a1 <image size>\n\n");
    return 1;
  }

  // Calculate image size.
  int width = atoi(argv[1]);
  int image_size = width*width;

  if (PRINT_TIME)
    printf("image size = %d\n", image_size);

  // Declare useful variables.
  struct timespec b_init, e_init, b_cpyHD, e_cpyHD, b_kernel, e_kernel, b_cpyDH, e_cpyDH, b_serial, e_serial;
  char *image, *sobel_image, *image_serial, *sobel_image_serial;
  FILE *output_fp, *output_serial_fp;

  // Start CUDA profiler.
  hipProfilerStart();

  // Initialize input data.
  b_init = now();
  srand(time(NULL));

  image = create_input(image_size);
  sobel_image = initialize_output(image_size);

  //image_serial = create_input(image_size);
  sobel_image_serial = initialize_output(image_size);

  int byte_size = image_size*sizeof(char);
  e_init = now();

  // Allocate device memory.
  char *d_image, *d_sobel_image;

  hipError_t err_image = hipMalloc((void**) &d_image, byte_size);
  if (PRINT_ERR)
    printf("CUDA malloc d_image: %s\n",hipGetErrorString(err_image));

  hipError_t err_sobel_image = hipMalloc((void**) &d_sobel_image, byte_size);
  if (PRINT_ERR)
    printf("CUDA malloc d_sobel_image: %s\n",hipGetErrorString(err_sobel_image));  

  // Copy data from host memory to device memory.
  b_cpyHD = now();
  err_image = hipMemcpy(d_image, image, byte_size, hipMemcpyHostToDevice);
  if (PRINT_ERR)
    printf("CUDA Memcpy image->d_image: %s\n",hipGetErrorString(err_image));

  err_sobel_image = hipMemcpy(d_sobel_image, sobel_image, byte_size, hipMemcpyHostToDevice);
  if (PRINT_ERR)
    printf("CUDA Memcpy sobel_image->d_sobel_image: %s\n",hipGetErrorString(err_sobel_image));
  e_cpyHD = now();

  // Launch the device kernel.
  b_kernel = now();
  VectorComputeSobel<<<width,width>>>(d_image, d_sobel_image, width);
  hipError_t err_VCS = hipDeviceSynchronize();
  e_kernel = now();

  // Copy data from device memory to host memory.
  b_cpyDH = now();
  err_sobel_image = hipMemcpy(sobel_image, d_sobel_image, byte_size, hipMemcpyDeviceToHost);
  if (PRINT_ERR)
    printf("CUDA Memcpy d_sobel_image->sobel_image: %s\n",hipGetErrorString(err_sobel_image));
  e_cpyDH = now();

  // Free the device memory.
  hipFree(d_image);
  hipFree(d_sobel_image);

  // Stop the CUDA profiler.
  hipProfilerStop();

  // Run the serial implementation of the 3x3 Sobel filter.
  b_serial = now();
  sobel_image_serial = sobel_serial(width, image);
  e_serial = now();
  
  // Store the output from the serial and GPU implementations of the 3x3 Sobel filter.
  //    Used for comparison to check for correctness of the code using the linux diff command.
  output_fp = fopen("output.txt", "w");
  output_serial_fp = fopen("output_serial.txt", "w");

  int i;
  for (i = 0; i < image_size; ++i) {
    fprintf(output_fp, "%d\n", (sobel_image[i] & 0x000000FF));
    fprintf(output_serial_fp, "%d\n", (sobel_image_serial[i] & 0x000000FF));
  }

  // Close the files and free the pointers.
  fclose(output_fp);
  fclose(output_serial_fp);

  free(image);
  free(sobel_image);

  //free(image_serial);
  free(sobel_image_serial);
  
  // Print elapsed times for the different portions of the code.
  if (PRINT_ERR) {
    printf("Time elapsed for initialization: %.8f sec\n", tdiff(b_init, e_init));
    printf("Time elapsed for memory copy from host to device: %.8f sec\n", tdiff(b_cpyHD, e_cpyHD));
  }
  if (PRINT_RESULT) {
    printf("Time elapsed for kernel: %.8f sec\n", tdiff(b_kernel, e_kernel));
//    printf("%.8f ", tdiff(b_kernel, e_kernel));
  }
  if (PRINT_ERR)
    printf("Time elapsed for memory copy from device to host: %.8f sec\n", tdiff(b_cpyDH, e_cpyDH));
  if (PRINT_RESULT) {
    printf("Time elapsed for serial execution: %.8f sec\n", tdiff(b_serial, e_serial));
//    printf("%.8f\n", tdiff(b_serial, e_serial));
  }

  return 0;

}
